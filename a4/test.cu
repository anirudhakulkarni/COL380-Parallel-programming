#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;

unordered_map<int, int> angle = {{-1, -45}, {0, 0}, {1, 45}};
unordered_map<int, float> cosine = {{-1, 0.70710}, {0, 1}, {1, 0.70710}};
unordered_map<int, float> sine = {{-1, -0.70710}, {0, 0}, {1, 0.70710}};

float matchTemplateGPU(vector<vector<vector<int>>> &queryImage, vector<vector<vector<int>>> &dataImage, int &i1, int &j1, int &k1)
{
	long long int temp = 0;
	int m = queryImage.size();
	for (int i = 0; i < queryImage.size(); i++)
	{
		for (int j = 0; j < queryImage[0].size(); j++)
		{
			for (int k = 0; k < 3; k++)
			{
				int i2 = i1 - int(i * cosine[k1] + j * sine[k1]), j2 = j1 + int(j * cosine[k1] - i * sine[k1]);
				temp += pow(queryImage[m - i - 1][j][k] - ((i2 >= 0 && i2 < dataImage.size() && j2 >= 0 && j2 < dataImage[0].size()) ? dataImage[i2][j2][k] : 0), 2);
			}
		}
	}
	return sqrt(((float)(temp)) / queryImage.size() / queryImage[0].size() / 3.0);
}

__global__ float matchTemplateGPU-kernel(vector<vector<vector<int>>> &queryImage, vector<vector<vector<int>>> &dataImage, int &i1, int &j1, int &k1)
{
  long long int temp = 0;
  int m = queryImage.size();
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int j=blockIdx.y*blockDim.y+threadIdx.y;
  int k=blockIdx.z*blockDim.z+threadIdx.z;
  if(i<m && j<queryImage[0].size() && k<3){
    int i2 = i1 - int(i * cosine[k1] + j * sine[k1]), j2 = j1 + int(j * cosine[k1] - i * sine[k1]);
    temp += pow(queryImage[m - i - 1][j][k] - ((i2 >= 0 && i2 < dataImage.size() && j2 >= 0 && j2 < dataImage[0].size()) ? dataImage[i2][j2][k] : 0), 2);
  }
  return sqrt(((float)(temp)) / queryImage.size() / queryImage[0].size() / 3.0);
}

float filterTemplate(vector<vector<long long int>> &queryImagePrefixSum, vector<vector<long long int>> &dataImagePrefixSum, int &i, int &j, int &k)
{
	int w = queryImagePrefixSum[0].size() - 1, h = queryImagePrefixSum.size() - 1;
	int i1, i2, j1, j2;
	// int i1 = i, j1 = j, i2 = i-int(0*cosine[k]+w*sine[k]), j2 = j+int(w*cosine[k]-0*sine[k]), i3 = i-int(h*cosine[k]+0*sine[k]), j3 = j+int(0*cosine[k]-h*sine[k]), i4 = i-int(h*cosine[k]+w*sine[k]), j4 = j+int(w*cosine[k]-h*sine[k]);
	if (k >= 0)
	{
		i1 = i, j1 = j + int(w * cosine[k] - 0 * sine[k]), i2 = i - int(h * cosine[k] + w * sine[k]), j2 = j + int(0 * cosine[k] - h * sine[k]);
	}
	else
	{
		i1 = i - int(0 * cosine[k] + w * sine[k]), j1 = j + int(w * cosine[k] - h * sine[k]), i2 = i - int(h * cosine[k] + 0 * sine[k]), j2 = j;
	}
	// if(i2>i1){
	// 	swap(i1, i2);
	// }
	// if(j2>j1){
	// 	swap(j1, j2);
	// }
	// i1 = max({i1, i2, i3, i4});
	// j1 = max({j1, j2, j3, j4});
	// i2 = min({i1, i2, i3, i4});
	// j2 = min({j1, j2, j3, j4});
	int w_d = j1 - j2 + 1, h_d = i1 - i2 + 1;
	i2 = max(i2, 0);
	j2 = max(j2, 0);
	i1 = min(i1, (int)dataImagePrefixSum.size() - 1);
	j1 = min(j1, (int)dataImagePrefixSum[0].size() - 1);
	// cout<<j2<<" "<<i2<<" "<<j1<<" "<<i1<<" "<<((float)(dataImagePrefixSum[i1][j1] - ((i2-1>=0)? dataImagePrefixSum[i2-1][j1]: 0) - ((j2-1>=0)? dataImagePrefixSum[i1][j2-1]: 0) + ((i2-1>=0 && j2-1>=0)? dataImagePrefixSum[i2-1][j2-1]: 0)))/w_d/h_d<<endl;
	return abs(((float)(dataImagePrefixSum[i1][j1] - ((i2 - 1 >= 0) ? dataImagePrefixSum[i2 - 1][j1] : 0) - ((j2 - 1 >= 0) ? dataImagePrefixSum[i1][j2 - 1] : 0) + ((i2 - 1 >= 0 && j2 - 1 >= 0) ? dataImagePrefixSum[i2 - 1][j2 - 1] : 0))) / w_d / h_d - ((float)(queryImagePrefixSum[w][h])) / (w + 1) / (h + 1));
}

void matchTemplateConfig(vector<vector<vector<int>>> &queryImage, vector<vector<long long int>> &queryImagePrefixSum, vector<vector<vector<int>>> &dataImage, vector<vector<long long int>> &dataImagePrefixSum, int &i, int &j, int &k, float &threshold1, float &threshold2, int &n, priority_queue<pair<float, vector<int>>> &top_n)
{
	float tempThreshold2 = filterTemplate(queryImagePrefixSum, dataImagePrefixSum, i, j, k);
	if (tempThreshold2 >= threshold2)
	{
		return;
	}
	cout << i << " " << j << " " << k << " " << tempThreshold2 << endl;
	// float tempThreshold1 = matchTemplateGPU(queryImage, dataImage, i, j, k);
  float tempThreshold1 = matchTemplateGPU-kernel(queryImage, dataImage, i, j, k);
	if (tempThreshold1 < threshold1)
	{
		top_n.push({tempThreshold1, {i, j, k}});
		while (top_n.size() > n)
		{
			top_n.pop();
		}
	}
}

void matchTemplate(vector<vector<vector<int>>> &queryImage, vector<vector<long long int>> &queryImagePrefixSum, vector<vector<vector<int>>> &dataImage, vector<vector<long long int>> &dataImagePrefixSum, float &threshold1, float &threshold2, int &n)
{
	priority_queue<pair<float, vector<int>>> top_n;
	for (int i = 0; i < dataImage.size(); i++)
	{
		for (int j = 0; j < dataImage[i].size(); j++)
		{
			for (int k = -1; k <= 1; k++)
			{
				matchTemplateConfig(queryImage, queryImagePrefixSum, dataImage, dataImagePrefixSum, i, j, k, threshold1, threshold2, n, top_n);
			}
		}
	}
	fstream fs("output.txt", fstream::out);
	int topn = top_n.size();
	vector<vector<int>> output(topn);
	while (top_n.size())
	{
		output[--topn] = top_n.top().second;
		top_n.pop();
	}
	int m = dataImage.size();
	for (int i = 0; i < output.size(); i++)
	{
		fs << m - output[i][0] - 1 << " " << output[i][1] << " " << angle[output[i][2]] << endl;
	}
	fs.close();
}

void readImage(string path_of_image, vector<vector<vector<int>>> &Image, vector<vector<long long int>> &ImagePrefixSum)
{
	fstream fs(path_of_image, fstream::in);
	int m, n;
	fs >> m >> n;
	vector<vector<vector<int>>> tempImage(m, vector<vector<int>>(n, vector<int>(3)));
	vector<vector<long long int>> tempImagePrefixSum(m, vector<long long int>(n));
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < n; j++)
		{
			int temp = 0;
			for (int k = 0; k < 3; k++)
			{
				fs >> tempImage[i][j][k];
				temp += tempImage[i][j][k];
			}
			tempImagePrefixSum[i][j] = ((j - 1 >= 0) ? tempImagePrefixSum[i][j - 1] : 0) + ((i - 1 >= 0) ? tempImagePrefixSum[i - 1][j] : 0) - ((i - 1 >= 0 && j - 1 >= 0) ? tempImagePrefixSum[i - 1][j - 1] : 0) + temp / 3.0;
		}
	}
	Image = tempImage;
	ImagePrefixSum = tempImagePrefixSum;
	fs.close();
}

int main(int argc, char const *argv[])
{
	auto begin = std::chrono::high_resolution_clock::now();
	string path_of_data_image = argv[1];
	string path_of_query_image = argv[2];
	float threshold1 = stof(argv[3]);
	float threshold2 = stof(argv[4]);
	int n = stoi(argv[5]);
	vector<vector<vector<int>>> dataImage, queryImage;
	vector<vector<long long int>> dataImagePrefixSum, queryImagePrefixSum;
	readImage(path_of_data_image, dataImage, dataImagePrefixSum);
	readImage(path_of_query_image, queryImage, queryImagePrefixSum);
	matchTemplate(queryImage, queryImagePrefixSum, dataImage, dataImagePrefixSum, threshold1, threshold2, n);
	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
	float duration = (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin)).count());
	cout << "Time taken " << duration << "ms" << endl;
	return 0;
}