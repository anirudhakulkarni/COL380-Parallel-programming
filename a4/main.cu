#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;

class compare
{
public:
    bool operator() (const pair<float, vector<int>> &a, const pair<float, vector<int>> &b){
        return a.first<b.first;
    }
};

#define pq priority_queue <pair<float, vector<int>>, vector<pair<float, vector<int>>>, compare >

int angle[3] = { -45,  0, 45 };
__device__ float  cosine[3] = { 0.70710678, 1, 0.70710678 };
__device__ float  sine[3] = { -0.70710678 , 0, 0.70710678 };

__global__ void matchTemplateGPU_kernel(int* queryImage, float *greyscaleAverageQ, int* dataImage, long long int *dataImagePrefixSum, int r_d, int c_d, int r_q, int c_q, float threshold1, float threshold2, float* anserarray){
    int absi = blockIdx.x * 256 + threadIdx.x;
    int i1 = absi / c_d;
    int j1 = absi % c_d;

    for (int k1 = 0;k1 < 3;k1++){
        float temp = 0;
        if (!(i1 < r_d && j1 < c_d && k1 < 3)) continue;
        anserarray[i1 * c_d * 3 + j1 * 3 + k1] = -1;
        int w = c_q - 1, h = r_q - 1;
        int i2 = i1-int(0*cosine[k1]+w*sine[k1]), j2 = j1+int(w*cosine[k1]-0*sine[k1]), i3 = i1-int(h*cosine[k1]+0*sine[k1]), j3 = j1+int(0*cosine[k1]-h*sine[k1]), i4 = i1-int(h*cosine[k1]+w*sine[k1]), j4 = j1+int(w*cosine[k1]-h*sine[k1]);
        int i_min = min(i1, min(i2, min(i3, i4))), i_max = max(i1, max(i2, max(i3, i4)));
        int j_min = min(j1, min(j2, min(j3, j4))), j_max = max(j1, max(j2, max(j3, j4)));
        int sizeD = (i_max-i_min+1)*(j_max-j_min+1);
        i_min = max(i_min, 0), i_max = min(i_max, r_d-1);
        j_min = max(j_min, 0), j_max = min(j_max, c_d-1);
        float greyscaleAverageD = float(dataImagePrefixSum[i_max*c_d+j_max] - ((i_min-1>=0)? dataImagePrefixSum[(i_min-1)*c_d+j_max]: 0) - ((j_min-1>=0)? dataImagePrefixSum[i_max*c_d+(j_min-1)]: 0) + ((i_min-1>=0 && j_min-1>=0)? dataImagePrefixSum[(i_min-1)*c_d+(j_min-1)]: 0))/sizeD;
        // float greyscaleAverageD=0.0;
        // for(int i=i_min;i<=i_max;i++){
        //     for(int j=j_min;j<=j_max;j++){
        //         int temp=0;
        //         for(int k_=0;k_<3;k_++){
        //             temp+=dataImage[(i) * 3 * c_d + j * 3 + k_];
                
        //         }
        //         greyscaleAverageD+=temp/3;        
        //     }                
        // }
        // greyscaleAverageD/=sizeD;
        float greyscale = abs(greyscaleAverageD - *greyscaleAverageQ);
        if (greyscale >= threshold2){
            continue;
        }
        for (int i = 0; i < r_q; i++){
            for (int j = 0; j < c_q; j++){
                float i_temp = i * cosine[k1] + j * sine[k1], j_temp = j * cosine[k1] - i * sine[k1];
                float i2 = i1 - i_temp, j2 = j1 + j_temp;
                int i2_min = min(i1 - floor(i_temp), i1 - ceil(i_temp)), i2_max = max(i1 - floor(i_temp), i1 - ceil(i_temp)), j2_min = min(j1 + floor(j_temp), j1 + ceil(j_temp)), j2_max = max(j1 + floor(j_temp), j1 + ceil(j_temp));
                float x = i2-i2_min, y = j2-j2_min;
                for (int k = 0; k < 3; k++){
                    float y1 = ((i2_min >= 0 && i2_min < r_d&& j2_min >= 0 && j2_min < c_d) ? dataImage[i2_min * 3 * c_d + j2_min * 3 + k] : 0)*(1-x)*(1-y) + ((i2_max >= 0 && i2_max < r_d&& j2_min >= 0 && j2_min < c_d) ? dataImage[i2_max * 3 * c_d + j2_min * 3 + k] : 0)*x*(1-y) + ((i2_min >= 0 && i2_min < r_d&& j2_max >= 0 && j2_max < c_d) ? dataImage[i2_min * 3 * c_d + j2_max * 3 + k] : 0)*(1-x)*y + ((i2_max >= 0 && i2_max < r_d&& j2_max >= 0 && j2_max < c_d) ? dataImage[i2_max * 3 * c_d + j2_max * 3 + k] : 0)*x*y;
                    float tempdhairya=float(queryImage[(r_q-i-1) * 3 * c_q + j * 3 + k]) - y1;
                    temp+=(tempdhairya*tempdhairya)/ (r_q * c_q * 3.0);
                    // temp += pow(float(queryImage[(r_q-i-1) * 3 * c_q + j * 3 + k]) - y1, 2) / (r_q * c_q * 3.0);

                }
            }
        }
        float ans;
        ans = sqrt(temp);
        if(ans<threshold1){
            anserarray[i1 * c_d * 3 + j1 * 3 + k1] = ans;
            // if(ans<10)
            // printf("%f, %f\n",ans,greyscale);
        }
    }
}


void matchTemplate_GPU(int* queryImage, float *greyscaleAverageQ, int* dataImage, long long int* dataImagePrefixSum, float& threshold1, float& threshold2, int& r_d, int& c_d, int& r_q, int& c_q, float* anserarray){
    matchTemplateGPU_kernel << <(r_d * c_d + 255) / 256, 256 >> > (queryImage, greyscaleAverageQ, dataImage, dataImagePrefixSum, r_d, c_d, r_q, c_q, threshold1, threshold2, anserarray);
}

void readImage_GPU(string path_of_image, int* &Image, long long int* &ImagePrefixSum, int &r,int &c){
  fstream fs(path_of_image, fstream::in);
  fs>>r>>c;
    Image=new int[r*c*3];
    ImagePrefixSum=new long long int[r*c];
  for(int i=0;i<r;i++){
    for(int j=0; j<c; j++){
        int temp = 0;
        for(int k=0; k<3; k++){
            fs>>Image[i*c*3+j*3+k];
            temp+=Image[i*c*3+j*3+k];
        }
        ImagePrefixSum[i*c+j] = ((j-1>=0)? ImagePrefixSum[i*c+(j-1)]: 0) + ((i-1>=0)? ImagePrefixSum[(i-1)*c+j]: 0) - ((i-1>=0 && j-1>=0)? ImagePrefixSum[(i-1)*c+(j-1)]: 0) + temp/3;
    }
  }
}

int main(int argc, char const* argv[])
{
    auto begin = std::chrono::high_resolution_clock::now();
    string path_of_data_image = argv[1];
    string path_of_query_image = argv[2];
    float threshold1 = stof(argv[3]);
    float threshold2 = stof(argv[4]);
    int n = stoi(argv[5]);
    int* dataImage; int* queryImage;
    long long int *dataImagePrefixSum;long long int *queryImagePrefixSum;
    int r_d, c_d, r_q, c_q; 
    readImage_GPU(path_of_data_image, dataImage, dataImagePrefixSum,r_d,c_d);
    readImage_GPU(path_of_query_image, queryImage, queryImagePrefixSum,r_q,c_q);
    float greyscaleAverageQTemp = float(queryImagePrefixSum[(r_q-1)*c_q+(c_q-1)])/(r_q*c_q);
    float *greyscaleAverageQ = &greyscaleAverageQTemp;
    
    float *greyscaleAverageQ_GPU;
    hipMalloc(&greyscaleAverageQ_GPU,sizeof(float));
    hipMemcpy(greyscaleAverageQ_GPU,greyscaleAverageQ,sizeof(float),hipMemcpyHostToDevice);
    
    long long int *dataImagePrefixSum_GPU;
    hipMalloc(&dataImagePrefixSum_GPU,r_d*c_d*sizeof(long long int));
    hipMemcpy(dataImagePrefixSum_GPU,dataImagePrefixSum,r_d*c_d*sizeof(long long int),hipMemcpyHostToDevice);

    int* dataImage_GPU;
    hipMalloc(&dataImage_GPU, r_d * c_d * 3 * sizeof(int));
    hipMemcpy(dataImage_GPU, dataImage, r_d * c_d * 3 * sizeof(int), hipMemcpyHostToDevice);
 
    int* queryImage_GPU;
    hipMalloc(&queryImage_GPU, r_q * c_q * 3 * sizeof(int));
    hipMemcpy(queryImage_GPU, queryImage, r_q * c_q * 3 * sizeof(int), hipMemcpyHostToDevice);
    
    cout << "Time taken for input reading:\t" << (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - begin)).count()) << "ms" << endl;

    float* anserarray = new float[r_d * c_d * 3];
    memset(anserarray, 0, sizeof(anserarray));

    float* anserarray_GPU;
    hipMalloc(&anserarray_GPU, r_d * c_d * 3 * sizeof(float));
    hipMemcpy(anserarray_GPU, anserarray, r_d * c_d * 3 * sizeof(float), hipMemcpyHostToDevice);
    cout << "Time taken for alllocate:\t" << (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - begin)).count()) << "ms" << endl;

    matchTemplate_GPU(queryImage_GPU, greyscaleAverageQ_GPU, dataImage_GPU, dataImagePrefixSum_GPU, threshold1, threshold2, r_d, c_d, r_q, c_q, anserarray_GPU);
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    hipDeviceSynchronize();
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    hipMemcpy(anserarray, anserarray_GPU, r_d * c_d * 3 * sizeof(float), hipMemcpyDeviceToHost);
    
    pq top_n;
    for (int i = 0; i < r_d ; i++) {
        for(int j=0; j<c_d; j++){
            for(int k=0; k<3; k++){
                if(anserarray[i * c_d * 3 + j * 3 + k]!=-1){
                    top_n.push(make_pair(anserarray[i * c_d * 3 + j * 3 + k], vector<int>({i, j, k})));
                    while(top_n.size()>n){
                        top_n.pop();
                    }
                }
            }
        }
    }
    vector<vector<int> > answer(top_n.size());
    // DEBUG
    vector<float> debug_answer(top_n.size());
    // DEBUG
    fstream fs_debug("debug.txt", fstream::out);

    for(int i=answer.size()-1; i>=0; i--){
        answer[i]=top_n.top().second;
        // DEBUG
        debug_answer[i]=top_n.top().first;
        top_n.pop();
    }
    fstream fs("output.txt", fstream::out);
    for(int i=0; i<answer.size(); i++){
        //change
        fs<<r_d-answer[i][0]-1<<" "<<answer[i][1]<<" "<<angle[answer[i][2]]<<endl;
        // DEBUG
        fs_debug<<debug_answer[i]<<" "<<endl;
    }
    // DEBUG
    fs_debug.close();
    fs.close();

    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
    float duration = (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin)).count());
    cout << "Time taken \t" << duration << "ms" << endl;
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    return 0;

}
