#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;
#define tempThreshold1 2;
#define tempThreshold2 2;

unordered_map<int, int> angle_ = {{-2, -90}, {-1, -45}, {0, 0}, {1, 45}, {2, 90}};
unordered_map<int, float> cos_ = {{-2, 0}, {-1, 0.414}, {0, 1}, {1, 0.414}, {2, 0}};
unordered_map<int, float> sin_ = {{-2, -1}, {-1, -0.414}, {0, 0}, {1, 0.414}, {2, 1}};

float matchTemplateGPU(vector<vector<vector<uint8_t>>> &queryImage, vector<vector<vector<uint8_t>>> &dataImage, int &i, int &j, int &k){

}

float filterTemplate(vector<vector<float>> &queryImagePrefixSum, vector<vector<float>> &dataImagePrefixSum, int &i, int &j, int &k){
	int w = queryImagePrefixSum[0].size(), h = queryImagePrefixSum.size();
	int i1 = i, j1 = j - floor(h*sin_[k]) -1, i2 = i + floor(w*cos_[k] + h*cos_[k]) -1, j2 = j + floor(w*cos_[k]) - 1;
	return abs((dataImagePrefixSum[max(i1, i2)][max(j1, j2)] - dataImagePrefixSum[max(min(i1, i2)-1, 0)][max(min(j1, j2)-1, 0)])/abs(i2-i1+1)/abs(j2-j1+1)-(queryImagePrefixSum[w-1][h-1]/(w)/(h)));
}

void matchTemplateConfig(vector<vector<vector<uint8_t>>> &queryImage, vector<vector<float>> &queryImagePrefixSum, vector<vector<vector<uint8_t>>> &dataImage, vector<vector<float>> &dataImagePrefixSum, int &i, int &j, int &k, float &threshold1, float &threshold2, int &n, priority_queue<pair<int, vector<int>> > &top_n){
	tempThreshold2 = filterTemplate(queryImagePrefixSum, dataImagePrefixSum, i, j, k);
	if(tempThreshold2>=threshold2){
		return;
	}
	tempThreshold1 = matchTemplateGPU(queryImage, dataImage, i, j, k);
	if(tempThreshold1<threshold1){
		top_n.push(make_pair(tempThreshold1, {i, j, k}));
		while(top_n.size()>n){
			top_n.pop();
		}
	}
}

void matchTemplate(vector<vector<vector<uint8_t>>> &queryImage, vector<vector<float>> &queryImagePrefixSum, vector<vector<vector<uint8_t>>> &dataImage, vector<vector<float>> &dataImagePrefixSum, float &threshold1, float &threshold2, int &n){
	priority_queue<pair<int, vector<int>> > top_n;
	for(int i=0; i<dataImage.size(); i++){
		for(int j=0; j<dataImage[i].size(); j++){
			for(int k=-2; k<=2; k++){
				matchTemplateConfig(queryImage, dataImage, i, j, k, threshold1, threshold2, k, top_n)
			}
		}
	}
	fstream fs("output.txt", fstream::out);
	int topn = top_n.size();
	vector<vector<int>> output(topn);
	while(top_n.size()){
		output[--topn] = top_n.top();
		top_n.pop();
	}
	int m = dataImage.size();
	for(int i=0; i<output.size(); i++){
		fs<<m-output[i][0]-1<<" "<<output[i][1]<<" "<<angle_[output[i][2]]<<endl;
	}
	fs.close();
}

void readImage(string path_of_image, vector<vector<vector<uint8_t>>> &Image, vector<vector<uint8_t>> &ImagePrefixSum){
	fstream fs(path_of_image, fstream::in);
	int m, n;
	fs>>m>>n;
	Image.resize(m);
	ImagePrefixSum.resize(m);
	vector<vector<vector<uint8_t>>> tempImage(m, vector<vector<uint8_t>>(n, vector<uint8_t>(3)))
	vector<vector<float>> tempImagePrefixSum(m, vector<vector<float>>(n, 0));
	for(int i=0; i<m; i++){
		for(int j=0; j<n; j++){
			float temp = 0.0;
			for(int k=0; k<3; k++){
				fs>>tempImage[i][j][k];
				temp+=tempImage[i][j][k];
			}
			tempImagePrefixSum[i][j] = tempImagePrefixSum[i][max(j-1, 0)] + tempImagePrefixSum[max(i-1, 0)][j] + temp/3.0;
		}
	}
	Image = tempImage;
	ImagePrefixSum = tempImagePrefixSum;
	fs.close();

}

int main(int argc, char const *argv[])
{
	auto begin = std::chrono::high_resolution_clock::now();
	string path_of_data_image = argv[1]; 
	string path_of_query_image = argv[2]; 
	float threshold1 = stof(argv[3]);
	float threshold2 = stof(argv[4]);
	int n = stoi(argv[5]);
	vector<vector<vector<uint8_t>>> dataImage, queryImage;
	vector<vector<float>> dataImagePrefixSum, queryImagePrefixSum;
	readImage(path_of_data_image, dataImage, dataImagePrefixSum);
	readImage(path_of_query_image, queryImage, queryImagePrefixSum);
	matchTemplate(queryImage, queryImagePrefixSum, dataImage, dataImagePrefixSum, threshold1, threshold2, n);
	auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
    float duration = (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin)).count());
    cout << "Time taken " << duration << "ms" << endl;
	return 0;
}