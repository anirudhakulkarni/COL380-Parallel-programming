#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
using namespace std;

// unordered_map<int, int> angle = {{-1, -45}, {0, 0}, {1, 45}};
// unordered_map<int, float> cosine = {{-1, 0.70710}, {0, 1}, {1, 0.70710}};
// unordered_map<int, float> sine = {{-1, -0.70710}, {0, 0}, {1, 0.70710}};

// __device__ int** angle_kernel={{-1, -45}, {0, 0}, {1, 45}};
// __device__ float**  cosine_kernel = {{-1, 0.70710}, {0, 1}, {1, 0.70710}};
// __device__ float**  sine_kernel = {{-1, -0.70710}, {0, 0}, {1, 0.70710}};
// 
__device__ int angle[3] = { -45,  0, 45 };
__device__ float  cosine[3] = { 0.70710, 1, 0.70710 };
__device__ float  sine[3] = { -0.70710 , 0, 0.70710 };

__global__ void matchTemplateGPU_kernel(int* queryImage, int* dataImage, int r_d, int c_d, int r_q, int c_q, float threshold1, float threshold2, int* anserarray)
{
    // int angle[3]={ -45,  0, 45};
    // float  cosine[3] = {0.70710, 1, 0.70710};
    // float  sine[3] = { -0.70710 , 0, 0.70710};
    // DEBUG:
  //   int i1=blockIdx.x*blockDim.x+threadIdx.x;
  //   int j1=blockIdx.y*blockDim.y+threadIdx.y;
  //   int k1=blockIdx.z*blockDim.z+threadIdx.z;
    int absi = blockIdx.x * 256 + threadIdx.x;
    int i1 = absi / c_d;
    int j1 = absi % c_d;
    //   if(i1==0)
    //   printf("%d, %d\n",i1,j1);
    // if (!(i1 < r_d && j1<c_d)){printf("i:%d, j:%d\n",i1,j1);return;}
    for (int k1 = 0;k1 < 3;k1++)
    {
        float temp = 0;
        if (!(i1 < r_d && j1 < c_d && k1 < 3))break;
        anserarray[i1 * c_d * 3 + j1 * 3 + k1] = 1;
        // if(r_d-i1-1==290 ){
        //   printf("%d, %d, %d\n",i1,j1,k1);
        // }
    // calculate greyscale difference
  //   float greyscale=0;
  //   if (greyscale >= threshold2)
  // 	{
  // 		return;
  // 	}
      // greyscale=queryImage[4];
  // 3*i*width+j*3+k
      // temp=temp+1;
        for (int i = 0; i < r_q; i++)
        {
            for (int j = 0; j < c_q; j++)
            {
                for (int k = 0; k < 3; k++)
                {

                    int i2 = i1 - int(i * cosine[k1] + j * sine[k1]), j2 = j1 + int(j * cosine[k1] - i * sine[k1]);
                    temp += pow(queryImage[(r_q - i - 1) * 3 * c_q + j * 3 + k] - ((i2 >= 0 && i2 < r_d&& j2 >= 0 && j2 < c_d) ? dataImage[i2 * 3 * c_d + j2 * 3 + k] : 0), 2);
                    // if(r_d-i1-1==290 && j1==119 &&k==2){
                    //     printf("I: %d,J: %d, K: %d, Temp: %d\n",i,j,k,temp);
                    //   }
                              // temp += pow(queryImage[((i2 >= 0 && i2 < r_d && j2 >= 0 && j2 < c_d) ? dataImage[i2*3*r_d+j2*3+k] : 0), 2);
                    // temp += pow(queryImage[(i )*3*c_q+j*3+k], 2);
                    // float f=queryImage[(r_q - i - 1)*3*c_q+j*3+k]);// - ((i2 >= 0 && i2 < r_d && j2 >= 0 && j2 < c_d) ? dataImage[i2*3*c_d+j2*3+k] : 0);
                    // float f=0.0001;
                    // printf("value: %f",temp);
                    // if(temp>10000)return;
                    // temp+=f;
                    // temp = temp+(f*f)/ r_q / c_q / 3.0;

                }
            }
        }
        float ans;
        //   printf("temp: %f\n",temp);

        ans = sqrt(((float)(temp)) / r_q / c_q / 3.0);
        //   printf("ans: %f",ans);
        if (ans < 30) {
            printf("ANSwer: I: %d, J: %d, k: %d, ans: %f\n", i1, j1, k1, ans);
            // cout<<i<<" "<<j<<" "<<k<<" "<<ans<<endl;
        }
    }
}


void matchTemplate_GPU(int* queryImage, int* dataImage, float& threshold1, float& threshold2, int& r_d, int& c_d, int& r_q, int& c_q, int* anserarray)
{
    // allocate memory for int*** queryImage
    cout << "r_d: " << r_d << " c_d: " << c_d << " r_q: " << r_q << " c_q: " << c_q << endl;
    //   matchTemplateGPU_kernel<<<dim3(r_d/256,c_d,3),256>>>(queryImage, dataImage,r_d,c_d,r_q,c_q,threshold1,threshold2);
    //   matchTemplateGPU_kernel<<<(r_d*c_d*3)/256,256>>>(queryImage, dataImage,r_d,c_d,r_q,c_q,threshold1,threshold2);
    // matchTemplateGPU_kernel<<<dim3(r_d,c_d),3>>>(queryImage, dataImage,r_d,c_d,r_q,c_q,threshold1,threshold2);
    matchTemplateGPU_kernel << <(r_d * c_d + 255) / 256, 256 >> > (queryImage, dataImage, r_d, c_d, r_q, c_q, threshold1, threshold2, anserarray);
}

void readImage_GPU(string path_of_image, int*& Image, int& r, int& c) {
    fstream fs(path_of_image, fstream::in);
    fs >> r >> c;
    Image = new int[r * c * 3];
    for (int i = 0;i < r * c * 3;i++) {
        fs >> Image[i];
    }
}
int main(int argc, char const* argv[])
{
    auto begin = std::chrono::high_resolution_clock::now();
    string path_of_data_image = argv[1];
    string path_of_query_image = argv[2];
    float threshold1 = stof(argv[3]);
    float threshold2 = stof(argv[4]);
    int n = stoi(argv[5]);
    int* dataImage; int* queryImage;
    // vector<vector<vector<int>>> dataImage, queryImage;
    // long long int **dataImagePrefixSum;long long int **queryImagePrefixSum;
    // vector<vector<long long int>> dataImagePrefixSum, queryImagePrefixSum;
    int r_d, c_d, r_q, c_q;
    readImage_GPU(path_of_data_image, dataImage, r_d, c_d);
    readImage_GPU(path_of_query_image, queryImage, r_q, c_q);



    int* dataImage_GPU;
    hipMalloc(&dataImage_GPU, r_d * c_d * 3 * sizeof(int));
    //   cout<<"allocagted\n";
    hipMemcpy(dataImage_GPU, dataImage, r_d * c_d * 3 * sizeof(int), hipMemcpyHostToDevice);
    //   cout<<"allocagted\n";
    int* queryImage_GPU;
    hipMalloc(&queryImage_GPU, r_q * c_q * 3 * sizeof(int));
    //   cout<<"allocagted\n";
    hipMemcpy(queryImage_GPU, queryImage, r_q * c_q * 3 * sizeof(int), hipMemcpyHostToDevice);
    //   cout<<"allocagted\n";
    cout << "Time taken for input reading:\t" << (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - begin)).count()) << "ms" << endl;

    int* anserarray = new int[r_d * c_d * 3];
    memset(anserarray, 0, sizeof(anserarray));

    int* anserarray_GPU;
    hipMalloc(&anserarray_GPU, r_d * c_d * 3 * sizeof(int));
    // hipMalloc(&anserarray, r_d * c_d * 3 * sizeof(int));
    // std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add
    hipMemcpy(anserarray_GPU, anserarray, r_d * c_d * 3 * sizeof(int), hipMemcpyHostToDevice);
    // memset(anserarray,0,sizeof(anserarray));
    cout << "Time taken for alllocate:\t" << (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now() - begin)).count()) << "ms" << endl;
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    matchTemplate_GPU(queryImage_GPU, dataImage_GPU, threshold1, threshold2, r_d, c_d, r_q, c_q, anserarray_GPU);
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add
    hipDeviceSynchronize();
    hipMemcpy(anserarray, anserarray_GPU, r_d * c_d * 3 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0;i < r_d * c_d * 3;i++) {
        if (!anserarray[i])cout << i << endl;
    }
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin);
    float duration = (1e-6 * (std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin)).count());
    cout << "Time taken \t" << duration << "ms" << endl;
    // hipError_t err = hipGetLastError();  // add
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl; // add

    return 0;
}